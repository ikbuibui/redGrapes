#include "hip/hip_runtime.h"
/* Copyright 2020-2024 Michael Sippel, Tapish Narwal
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */

#include <redGrapes/SchedulerDescription.hpp>
#include <redGrapes/dispatch/cuda/cuda_task_properties.hpp>
#include <redGrapes/dispatch/cuda/cuda_worker.hpp>
#include <redGrapes/redGrapes.hpp>
#include <redGrapes/resource/fieldresource.hpp>
#include <redGrapes/resource/ioresource.hpp>
#include <redGrapes/scheduler/cuda_thread_scheduler.hpp>
#include <redGrapes/scheduler/pool_scheduler.hpp>
#include <redGrapes/task/property/resource.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pngwriter.h>

#include <chrono>
#include <functional>
#include <iomanip>
#include <iostream>

struct Color
{
    float r, g, b;
};

__global__ void hello_world()
{
}

__global__ void mandelbrot(
    double begin_x,
    double end_x,
    double begin_y,
    double end_y,
    int buffer_width,
    int buffer_height,
    Color* out)
{
    int xi = blockIdx.x * blockDim.x + threadIdx.x;
    int yi = blockIdx.y * blockDim.y + threadIdx.y;
    int index = xi + yi * buffer_width;
    double xf = begin_x + (end_x - begin_x) * double(xi) / double(buffer_width);
    double yf = begin_y + (end_y - begin_y) * double(yi) / double(buffer_height);

    double z_re = 0.0;
    double z_im = 0.0;
    int i;
    for(i = 0; i < 1000 && (z_re * z_re + z_im * z_im) < 4; ++i)
    {
        double new_z_re = z_re * z_re - z_im * z_im + xf;
        z_im = 2 * z_re * z_im + yf;
        z_re = new_z_re;
    }

    if(i == 1000)
        out[index] = Color{0.0, 0.0, 0.0};
    else
        out[index] = Color{cosf(float(i) / 7.0), cosf(2.0 + float(i) / 11.0), cosf(4.0 + float(i) / 13.0)};
}

struct CudaTag
{
};

using RGTask = redGrapes::Task<redGrapes::dispatch::cuda::CudaTaskProperties>;

int main()
{
    spdlog::set_level(spdlog::level::trace);
    spdlog::set_pattern("[thread %t] %^[%l]%$ %v");

    auto rg = redGrapes::init<redGrapes::dispatch::cuda::CudaTaskProperties>(
        redGrapes::SchedulerDescription(
            std::make_shared<redGrapes::scheduler::CudaThreadScheduler<RGTask>>(2),
            CudaTag{}),
        redGrapes::SchedulerDescription(
            std::make_shared<redGrapes::scheduler::PoolScheduler<redGrapes::dispatch::thread::DefaultWorker<RGTask>>>(
                4),
            redGrapes::DefaultTag{}));

    auto& cudaSched = rg.getScheduler<CudaTag>();

    double mid_x = 0.41820187155955555;
    double mid_y = 0.32743154895555555;

    size_t width = 4096;
    size_t height = 4096;
    size_t area = width * height;

    redGrapes::IOResource<Color*> host_buffer;
    redGrapes::IOResource<Color*> device_buffer;

    rg.emplace_task(
        [area](auto host_buffer)
        {
            void* ptr;
            hipHostMalloc(&ptr, area * sizeof(Color));
            *host_buffer = (Color*) ptr;
        },
        host_buffer.write());

    rg.emplace_task(
        [area](auto device_buffer)
        {
            void* ptr;
            hipMalloc(&ptr, area * sizeof(Color));
            *device_buffer = (Color*) ptr;
        },
        device_buffer.write());

    // warmup cuda
    // hello_world<<< 1, 1, 0, 0 >>>();
    // hipMemcpy(*host_buffer, *device_buffer, sizeof(Color), hipMemcpyDeviceToHost);

    auto t1 = std::chrono::high_resolution_clock::now();

    float w = 1.0;
    for(int i = 0; i < 10; ++i)
    {
        w *= 0.75;
        /*
         * calculate picture
         */
        rg.emplace_task<CudaTag>(
            [width, height, area, i, mid_x, mid_y, w, &cudaSched](auto device_buffer)
            {
                double begin_x = mid_x - w;
                double end_x = mid_x + w;
                double begin_y = mid_y - w;
                double end_y = mid_y + w;

                dim3 threadsPerBlock(8, 8);
                dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);

                auto current_stream = cudaSched.getCudaStream();
                mandelbrot<<<numBlocks, threadsPerBlock, 0, current_stream>>>(
                    begin_x,
                    end_x,
                    begin_y,
                    end_y,
                    width,
                    height,
                    *device_buffer);
                std::cout << "launched kernel to stream " << current_stream << std::endl;
            },
            device_buffer.write());

        /*
         * copy data
         */
        rg.emplace_task<CudaTag>(
            [area, &cudaSched](auto host_buffer, auto device_buffer)
            {
                auto current_stream = cudaSched.getCudaStream();
                hipMemcpyAsync(
                    *host_buffer,
                    *device_buffer,
                    area * sizeof(Color),
                    hipMemcpyDeviceToHost,
                    current_stream);
                std::cout << "launched memcpy to stream " << current_stream << std::endl;
            },
            host_buffer.write(),
            device_buffer.read());
        ;

        /*
         * write png
         */
        rg.emplace_task(
            [width, height, i](auto host_buffer)
            {
                std::stringstream step;
                step << std::setw(6) << std::setfill('0') << i;

                std::string filename("mandelbrot_" + step.str() + ".png");
                pngwriter png(width, height, 0, filename.c_str());
                png.setcompressionlevel(9);

                for(size_t y = 0; y < height; ++y)
                {
                    for(size_t x = 0; x < width; ++x)
                    {
                        auto& color = (*host_buffer)[x + y * width];
                        png.plot(x + 1, height - y, color.r, color.g, color.b);
                    }
                }

                png.close();
                std::cout << "wrote png" << std::endl;
            },
            host_buffer.read());
    }

    rg.emplace_task([]([[maybe_unused]] auto b) {}, host_buffer.write()).get();

    auto t2 = std::chrono::high_resolution_clock::now();
    std::cout << "runtime: " << std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count() << " μs"
              << std::endl;

    /*
     * cleanup
     */
    rg.emplace_task<CudaTag>([](auto host_buffer) { hipHostFree(*host_buffer); }, host_buffer.write());

    rg.emplace_task<CudaTag>([](auto device_buffer) { hipFree(*device_buffer); }, device_buffer.write());
}
